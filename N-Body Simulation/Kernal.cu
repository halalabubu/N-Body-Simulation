#include "hip/hip_runtime.h"
#include "Kernal.cuh"


//int index = blockIdx.x * blockDim.x + threadIdx.x;
//int stride = blockDim.x * gridDim.x;
//for (int i = index; i < n; i += stride)
  //  int x;

__global__ void setTextureColor(int n, sf::Uint8* pixels)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		pixels[i] = 0x2F;
}




//place particles into array
__global__ void placeIntoPixel(int n, int pixelLength, Particle* particle, Pixel* pixel) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)

	{

		int indexX = particle[i].pos.x;
		int indexY = particle[i].pos.y;

		//TODO allow multiple particles in the same place
		//pixel[indexX + indexY * pixelLength].particles = new Particle;
		pixel[indexX + indexY * pixelLength].particles = &particle[i];
		pixel[indexX + indexY * pixelLength].totalMass += particle[i].mass;

	}




}
//place particles into array
__global__ void placeIntoGrid(int n, int gridLength, float gridSize, Particle* particle, Grid* grid) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{

		int indexX = particle[i].pos.x / gridSize;
		int indexY = particle[i].pos.y / gridSize;

		//TODO mass is assumed to be 1 fix later?
		//TODO allow multiple particles in the same place
		//grid[indexX + indexY * gridLength].com.x = particle[i];
		grid[indexX + indexY * gridLength].totalMass += particle[i].mass;



	}
}

__global__ void updateTexture(int n, sf::Uint8* pixels, Pixel* pixel) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{
		if (pixel[i].totalMass > 0)
		{
			//make red
			pixels[i * 4] = 0xFF;	//R
			pixels[i * 4+1] = 0xFF;	//G
			pixels[i * 4+2] = 0xFF;	//B
			pixels[i * 4+3] = 0xFF;	//A
		}
		else
		{
			//make black
			pixels[i * 4] = 0x0F;
			pixels[i * 4+1] = 0x0F;
			pixels[i * 4+2] = 0x0F;
			pixels[i * 4+3] = 0xFF;//all alpha is always 0xFF
		}

	}

}
