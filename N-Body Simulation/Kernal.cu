#include "hip/hip_runtime.h"
#include "Kernal.cuh"

//int index = blockIdx.x * blockDim.x + threadIdx.x;
//int stride = blockDim.x * gridDim.x;
//for (int i = index; i < n; i += stride)
  //  int x;


__global__ void buildTree(
	Particle* particles, Node* node, NodeList* nList, int currentLevel, int maxlevels, float width, float xmin, float ymin) {
	//we will only ever have 1 thread per funtion call :( but will have thousands of calls

	node->mins.x = xmin;
	node->mins.y = ymin;
	node->width = width;

	// if levels = 0 return or 1 particle set mass and com return
	//TODO
	//edit this if we want to set min parts before sub

	//node->quadrants = new Node[4];

	if (node->startIndex == node->endIndex)
	{
		node->com = particles[node->startIndex].pos;
		node->totalMass = particles[node->startIndex].mass;
		return;
	}
	if (currentLevel == maxlevels)
	{
		float2 comT; comT.x = 0; comT.y = 0;
		float totalmass = 0;
		for (size_t i = node->startIndex; i < node->endIndex; i++)
		{
			comT.x = particles[i].pos.x * particles[i].mass;
			comT.y = particles[i].pos.y * particles[i].mass;
			totalmass += particles[i].mass;
		}
		node->com.x = comT.x / totalmass;
		node->com.y = comT.y / totalmass;
		node->totalMass = totalmass;
		return;
	}
	//we need to init head node values
	Particle temp;

	int qEnd0;
	int qEnd2;
	int qEnd3 = node->endIndex;//the end node is the last node
	int qEnd1 = node->endIndex;

	int partcount[4] = { 0,0,0,0 };

	// sort quad 0,1 on left and 2,3 on right
	// also gives us qEnd1 value;
	for (size_t i = node->startIndex; i < qEnd1; i)
	{
		if (particles[i].pos.y > ymin + width / 2)
		{
			temp = particles[qEnd1];
			particles[qEnd1] = particles[i];
			particles[i] = temp;
			qEnd1--;
		}
		else
			i++;
	}

	// calc qEnd0
	qEnd0 = qEnd1;
	for (size_t i = node->startIndex; i < qEnd0; i)
	{
		if (particles[i].pos.x >= xmin + width / 2)
		{
			temp = particles[qEnd0];
			particles[qEnd0] = particles[i];
			particles[i] = temp;
			qEnd0--;
			partcount[1]++;
		}
		else
		{
			i++;
			partcount[0]++;
		}
	}

	//do the same for bot side
	// calc qEnd2
	qEnd2 = qEnd3;
	for (size_t i = qEnd1; i < qEnd2; i)
	{
		if (particles[i].pos.x >= xmin + width / 2)
		{
			temp = particles[qEnd2];
			particles[qEnd2] = particles[i];
			particles[i] = temp;
			qEnd2--;
			partcount[3]++;
		}
		else
		{
			i++;
			partcount[2]++;
		}
	}

	//hipMallocManaged(node->quadrants, 4*sizeof(Node));
	//quadrants = new Node[4];











	//set values for the 4 new quadrants
	int tindex = node->startIndex;
	for (size_t i = 0; i < 4; i++)
	{
		if (partcount[i] > 0)
		{
			node->quadrants[i]->startIndex = tindex;
			node->quadrants[i]->endIndex = node->quadrants[i]->startIndex + partcount[i] - 1;
			tindex = node->quadrants[i]->endIndex + 1;


			//quadrants[i].startIndex = tindex;
			//quadrants[i].endIndex = quadrants[i].startIndex + partcount[i] - 1;
			//tindex = quadrants[i].endIndex;
		}
		else
			node->quadrants[i]->totalMass == 0;
	}


	//before new function calls
	currentLevel++;
	//build tree call here

	//buildTree <<<1, 1 >>> (particles,&nList->level0,nList, CURRENT_LEVEL, LEVELS, GRIDSIZE, 0, 0);

	buildTree << <1, 1 >> > (particles, node->quadrants[0], nList, currentLevel, maxlevels, width / 2, xmin, ymin);
	buildTree << <1, 1 >> > (particles, node->quadrants[1], nList, currentLevel, maxlevels, width / 2, xmin + width / 2, ymin);
	buildTree << <1, 1 >> > (particles, node->quadrants[2], nList, currentLevel, maxlevels, width / 2, xmin, ymin + width / 2);
	buildTree << <1, 1 >> > (particles, node->quadrants[3], nList, currentLevel, maxlevels, width / 2, xmin + width / 2, ymin + width / 2);

	// wait for these to finish
	hipDeviceSynchronize();


	//calc mass and com here
	node->totalMass = 0;
	float totalX = 0, totalY = 0;

	for (size_t i = 0; i < 4; i++)
	{
		if (node->quadrants[i] != nullptr)
		{
			node->totalMass += node->quadrants[i]->totalMass;
			totalX += node->quadrants[i]->com.x * node->quadrants[i]->totalMass;
			totalY += node->quadrants[i]->com.y * node->quadrants[i]->totalMass;
		}
	
	}
	node->com.x = totalX / node->totalMass;
	node->com.y = totalY / node->totalMass;

}

//__global__ void

//void calcAccel();
//void calcVelocity

