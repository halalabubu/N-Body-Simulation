#include "hip/hip_runtime.h"
#include "Kernal.cuh"



__global__ void buildTree(
	Particle* particles, Node* node, NodeList* nList, int currentLevel, int maxlevels, float width, float xmin, float ymin) {
	//we will only ever have 1 thread per funtion call :( but will have thousands of calls

	node->mins.x = xmin;
	node->mins.y = ymin;
	node->width = width;
	node->isLeaf = false;
	// if levels = 0 return or 1 particle set mass and com return
	//TODO
	//edit this if we want to set min parts before sub

	//node->quadrants = new Node[4];
	int minParts = 10;
	//if (node->startIndex == node->endIndex)
	if (node->endIndex - node->startIndex < minParts)
	{
		float2 comT; comT.x = 0; comT.y = 0;
		float totalmass = 0;
		for (size_t i = node->startIndex; i <= node->endIndex; i++)
		{
			comT.x = particles[i].pos.x * particles[i].mass;
			comT.y = particles[i].pos.y * particles[i].mass;
			totalmass += particles[i].mass;
			particles[i].node = node;
		}
		node->com.x = comT.x / totalmass;
		node->com.y = comT.y / totalmass;
		node->totalMass = totalmass;
		for (size_t i = 0; i < 4; i++) {
			node->quadrants[i]->totalMass = 0;
			node->quadrants[i]->isLeaf = false;
		}
		node->isLeaf = true;
		return;
	}
	if (currentLevel == maxlevels)
	{
		float2 comT; comT.x = 0; comT.y = 0;
		float totalmass = 0;
		for (size_t i = node->startIndex; i <= node->endIndex; i++)
		{
			comT.x = particles[i].pos.x * particles[i].mass;
			comT.y = particles[i].pos.y * particles[i].mass;
			totalmass += particles[i].mass;
			particles[i].node = node;
		}
		node->com.x = comT.x / totalmass;
		node->com.y = comT.y / totalmass;
		node->totalMass = totalmass;
		node->isLeaf = true;
		return;
	}
	//we need to init head node values
	Particle temp;

	int qEnd0;
	int qEnd2;
	int qEnd3 = node->endIndex;//the end node is the last node
	int qEnd1 = node->endIndex;

	int partcount[4] = { 0,0,0,0 };
	int topCount = 0;
	int botCount = 0;

	// sort quad 0,1 on left and 2,3 on right
	// also gives us qEnd1 value;
	for (int i = node->startIndex; i <= qEnd1; i)
	{
		if (particles[i].pos.y > ymin + width / 2)
		{
			temp = particles[qEnd1];
			particles[qEnd1] = particles[i];
			particles[i] = temp;
			qEnd1--;
			botCount++;
		}
		else {
			i++;
			topCount++;
		}
	}

	// calc qEnd0
	qEnd0 = node->startIndex + topCount - 1;
	if (topCount > 0)
	{

		for (int i = node->startIndex; i <= qEnd0; i)
		{
			if (particles[i].pos.x >= xmin + width / 2)
			{
				temp = particles[qEnd0];
				particles[qEnd0] = particles[i];
				particles[i] = temp;
				qEnd0--;
				partcount[1]++;
			}
			else
			{
				i++;
				partcount[0]++;
			}
		}
	}


	//do the same for bot side
	// calc qEnd2
	qEnd2 = node->endIndex;
	qEnd1 = node->startIndex + topCount;


	if (botCount > 0)
	{
		for (int i = qEnd1; i <= qEnd2; i)
		{
			if (particles[i].pos.x >= xmin + width / 2)
			{
				temp = particles[qEnd2];
				particles[qEnd2] = particles[i];
				particles[i] = temp;
				qEnd2--;
				partcount[3]++;
			}
			else
			{
				i++;
				partcount[2]++;
			}
		}
	}



	//hipMallocManaged(node->quadrants, 4*sizeof(Node));
	//quadrants = new Node[4];











	//set values for the 4 new quadrants
	int tindex = node->startIndex;
	for (size_t i = 0; i < 4; i++)
	{
		if (partcount[i] > 0)
		{
			node->quadrants[i]->startIndex = tindex;
			node->quadrants[i]->endIndex = node->quadrants[i]->startIndex + partcount[i] - 1;
			tindex = node->quadrants[i]->endIndex + 1;


			//quadrants[i].startIndex = tindex;
			//quadrants[i].endIndex = quadrants[i].startIndex + partcount[i] - 1;
			//tindex = quadrants[i].endIndex;
		}
		else
			node->quadrants[i]->totalMass = 0;
	}


	//before new function calls
	currentLevel++;
	//build tree call here

	//buildTree <<<1, 1 >>> (particles,&nList->level0,nList, CURRENT_LEVEL, LEVELS, GRIDSIZE, 0, 0);



	if (partcount[0] > 0)
		buildTree << <1, 1 >> > (particles, node->quadrants[0], nList, currentLevel, maxlevels, width / 2, xmin, ymin);
	else
		node->quadrants[0]->totalMass = 0;

	if (partcount[1] > 0)
		buildTree << <1, 1 >> > (particles, node->quadrants[1], nList, currentLevel, maxlevels, width / 2, xmin + width / 2, ymin);
	else
		node->quadrants[1]->totalMass = 0;

	if (partcount[2] > 0)
		buildTree << <1, 1 >> > (particles, node->quadrants[2], nList, currentLevel, maxlevels, width / 2, xmin, ymin + width / 2);
	else
		node->quadrants[2]->totalMass = 0;

	if (partcount[3] > 0)
		buildTree << <1, 1 >> > (particles, node->quadrants[3], nList, currentLevel, maxlevels, width / 2, xmin + width / 2, ymin + width / 2);
	else
		node->quadrants[3]->totalMass = 0;





	// wait for these to finish
	hipDeviceSynchronize();


	//calc mass and com here
	node->totalMass = 0;
	float totalX = 0, totalY = 0;

	for (size_t i = 0; i < 4; i++)
	{
		//if (node->quadrants[i] != nullptr)
		//{
		node->totalMass += node->quadrants[i]->totalMass;
		totalX += node->quadrants[i]->com.x * node->quadrants[i]->totalMass;
		totalY += node->quadrants[i]->com.y * node->quadrants[i]->totalMass;
		//}

	}
	node->com.x = totalX / node->totalMass;
	node->com.y = totalY / node->totalMass;

}




//int index = blockIdx.x * blockDim.x + threadIdx.x;
//int stride = blockDim.x * gridDim.x;
//for (int i = index; i < n; i += stride)
//int x;

// we dont need to sqrt since the accel form requires r^2 (2D)
__device__ float getDistance(float2& a, float2& b) {
	return  pow(b.x - a.x, 2) + pow(b.y - a.y, 2);
}
__device__ float2 getAccel(float2& a, float2& b, float& mass) {
	float2 accel;
	float dist = getDistance(a, b);
	accel.x = mass * (b.x - a.x) / (dist + 0.00001f);
	accel.y = mass * (b.y - a.y) / (dist + 0.00001f);

	return  accel;
}

__device__ float2 getallAccel(float2& pPos, Node* node, Particle* particles)
{
	float2 accel, temp;
	accel.x = 0.0f;
	accel.y = 0.0f;
	for (size_t i = node->startIndex; i <= node->endIndex; i++)
	{

			temp = getAccel(pPos, particles[i].pos, particles[i].mass);
			accel.x += temp.x;
			accel.y += temp.y;
		

	}

	return accel;
}




__device__ float2 checkChildren(float2& pPos, Node* currentNode, Particle* particles) {
	float2 accel;
	float2 temp;
	accel.x = 0;
	accel.y = 0;
	for (size_t i = 0; i < 4; i++)
	{
		if (currentNode->quadrants[i]->totalMass > 0)
		{
			float dist = getDistance(pPos, currentNode->quadrants[i]->com);
			if (dist > 22500)
			{
				temp = getAccel(pPos, currentNode->quadrants[i]->com, currentNode->quadrants[i]->totalMass);
				accel.x += temp.x;
				accel.y += temp.y;
			}
			else if (currentNode->quadrants[i]->isLeaf == true)
			{
				temp = getallAccel(pPos, currentNode->quadrants[i], particles);
				accel.x += temp.x;
				accel.y += temp.y;
			}
			else
			{
				temp = checkChildren(pPos, currentNode->quadrants[i], particles);
				accel.x += temp.x;
				accel.y += temp.y;
			}

		}


	}
	return accel;
}

__device__ float2 traverseNode(float2& pPos, Node* currentNode, Particle* particles) {
	//check for if at top
	float2 accel;
	float2 temp;
	accel.x = 0;
	accel.y = 0;
	Node* prevNode;
	//TODO
	//calc for original node parts
	temp = getallAccel(pPos, currentNode, particles);
	accel.x += temp.x;
	accel.y += temp.y;


	while (currentNode->parent != nullptr)
	{
		//move up
		//traverseNode(pPos, currentNode->parent, currentNode);
		prevNode = currentNode;
		currentNode = currentNode->parent;
		//look at children
		for (size_t i = 0; i < 4; i++)
		{
			if (currentNode->quadrants[i] != prevNode)
			{
				temp = checkChildren(pPos, currentNode->quadrants[i], particles);
				accel.x += temp.x;
				accel.y += temp.y;
			}

		}



	}


	return accel;
}
__global__ void setVelSetPos(Particle* particles, NodeList* nlist, float delta) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	float2 temp;
	for (int i = index; i <= nlist->level0.endIndex; i += stride)
	{
		temp = traverseNode(particles[i].pos, particles[i].node, particles);
		particles[i].velocity.x = particles[i].velocity.x + 0.05f * temp.x;
		particles[i].velocity.y = particles[i].velocity.y + 0.05f * temp.y;
		particles[i].pos.x = particles[i].pos.x + particles[i].velocity.x;
		particles[i].pos.y = particles[i].pos.y + particles[i].velocity.y;
	}





}

//void calcAccel();
//void calcVelocity

//__global__ void naive(Particle* particles, NodeList* nlist)